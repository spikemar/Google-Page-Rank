#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <errno.h>
#include <limits.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define GIG 1000000000
#define CPG 2.4           // Cycles per GHz -- Adjust to your computer
#define GPU_BLOCK_SIZE 128

typedef float pr_type_t;

typedef struct ad_vert {
	long vertex_num;
	struct ad_vert *next;
}adj_vert_t;

typedef struct {
	pr_type_t curr_page_rank;
	pr_type_t next_page_rank;
	long num_adj_nodes;
	adj_vert_t *last_node_addr;
	void *next;
}vertex_t;

typedef struct {
	long edge_index;
	char is_leaf;
}compact_adj_node_t;

typedef struct {
	pr_type_t next;
	pr_type_t curr;
}p_rank_struct_t;

pr_type_t epsilon;
pr_type_t rand_hop = 0.15;
__device__ pr_type_t d_rand_hop = 0.15;

#define GRAPH_FILE_SEPERATOR " ,;"
#define MAX_LINE_LEN 100
#define RAND_HOP_LIKELIHOOD(r_hop_prob, nvert) ((r_hop_prob) / (nvert))
#define TRAV_LIKELIHOOD(r_hop_prob, p, index, num_adj_nodes) ((1 - (r_hop_prob)) * (p)[index].curr / num_adj_nodes)
#define TRAV_LIKELIHOOD_LEAF(r_hop_prob, p, index, num_vertices) ((1 - (r_hop_prob)) * (p)[index].curr / (num_vertices - 1))

struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}

long string_to_long(char *str)
{
	long val;
	char *endptr;
	errno = 0;
    val = strtol(str, &endptr, 10);
	if((errno == ERANGE && (val == LONG_MAX || val == LONG_MIN)) || (errno != 0 && val == 0) || (endptr == str)) 
	{
		perror("Error while converting string to long value");
		val = -1;
	}
	return val;
}

void initialize_vertices(vertex_t *g, long num_vertices)
{
	long i;
	for(i = 0;i < num_vertices;i++)
	{
		g[i].curr_page_rank = 1 / (pr_type_t)num_vertices;
		g[i].next_page_rank = RAND_HOP_LIKELIHOOD(rand_hop, num_vertices);
		g[i].num_adj_nodes = 0;
		g[i].last_node_addr = NULL;
		g[i].next = NULL;
	}
}

int append_node(vertex_t *g, long parent_vertex, long child_vertex, long num_verts, long *num_edges)
{
	if(parent_vertex >= num_verts || child_vertex >= num_verts)
	{
		printf("Invalid arguments\n");
		return -1;
	}
	adj_vert_t *ptr = (adj_vert_t *)malloc(sizeof(adj_vert_t));
	ptr->vertex_num = child_vertex;
	ptr->next = NULL;
	if(g[parent_vertex].next == NULL)
	{
		g[parent_vertex].next = ptr;
		g[parent_vertex].last_node_addr = ptr;
	}
	else
	{
		g[parent_vertex].last_node_addr->next = ptr;
		g[parent_vertex].last_node_addr = ptr;
	}
	g[parent_vertex].num_adj_nodes++;
	(*num_edges)++;
	return 0;
}

void calc_bfs_pg_rank_serial(compact_adj_node_t *vertex_array,long *edge_array,char *frontier_array,char *visited_array,p_rank_struct_t *p_rank_array,long num_vertices,long num_edges,long *num_front,long i)
{
	long j, term_ind;
	pr_type_t p_rank_val;
	if(frontier_array[i])
	{
		frontier_array[i] = 0;
		(*num_front)--;
		visited_array[i] = 1;
		term_ind = (i == num_vertices - 1) ? num_edges : vertex_array[i + 1].edge_index;
		p_rank_val = TRAV_LIKELIHOOD(rand_hop,p_rank_array,i,(term_ind - vertex_array[i].edge_index));
		if(!vertex_array[i].is_leaf)
		{
			for(j = vertex_array[i].edge_index;j < term_ind;j++)
			{
				p_rank_array[edge_array[j]].next += p_rank_val;
				if(!visited_array[edge_array[j]])
				{
					if(!frontier_array[edge_array[j]])
					{
						(*num_front)++;
						frontier_array[edge_array[j]] = 1;
					}
				}
			}
		}
		else
		{
			for(j = 0;j < num_vertices && j != i;j++)
				p_rank_array[j].next += TRAV_LIKELIHOOD_LEAF(rand_hop,p_rank_array,i,num_vertices);
		}	
	}
}

__global__ void calc_bfs_pg_rank_cuda(compact_adj_node_t *vertex_array,long *edge_array,p_rank_struct_t *p_rank_array,long num_vertices,long num_edges)
{
	long i, j, term_ind;
	pr_type_t p_rank_val;

	i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < num_vertices)
	{
		term_ind = (i == num_vertices - 1) ? num_edges : vertex_array[i + 1].edge_index;
		if(!vertex_array[i].is_leaf)
		{
			p_rank_val = TRAV_LIKELIHOOD(d_rand_hop,p_rank_array,i,(term_ind - vertex_array[i].edge_index));
			for(j = vertex_array[i].edge_index;j < term_ind;j++)
				atomicAdd(&p_rank_array[edge_array[j]].next,p_rank_val);
		}
		else
		{
			p_rank_val = TRAV_LIKELIHOOD_LEAF(d_rand_hop,p_rank_array,i,num_vertices);
			for(j = 0;j < num_vertices;j++)
			{
				if(j != i)
					atomicAdd(&p_rank_array[j].next,p_rank_val);
			}
		}
	}	
}

__global__ void update_pr(p_rank_struct_t *p_rank_array, pr_type_t *pr_diff_max, long num_vertices)
{
	long i;
	pr_type_t curr_diff;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < num_vertices)
	{
		curr_diff = fabsf(p_rank_array[i].next - p_rank_array[i].curr);
		atomicAdd(pr_diff_max,curr_diff);	
		p_rank_array[i].curr = p_rank_array[i].next;
		p_rank_array[i].next = RAND_HOP_LIKELIHOOD(d_rand_hop,num_vertices);
	}
}

void print_converged_pr_vals(p_rank_struct_t *p_rank, long num_vertices)
{
	long i;
	pr_type_t sum=0;
	for(i = 0;i < num_vertices;i++){
		printf("Converged page rank for node %lu : %.10f\n",i,p_rank[i].curr);
		sum += p_rank[i].curr;
	}
	printf("Sum is %f\n",sum);
}

int main(int argc, char *argv[])
{
	hipEvent_t start, stop;
	float elapsed_gpu;

	long i,j;
	FILE *file;
	char *token1, *token2;
	char line[MAX_LINE_LEN];
	pr_type_t *d_pr_diff;
	pr_type_t pr_diff;
	long num_vertices = 0;
	long pnode, cnode;
	long iterations=0;
	vertex_t *graph;
	compact_adj_node_t *vertex_array, *d_vertex_array;
	long *edge_array, *d_edge_array;
	p_rank_struct_t *p_rank_array, *d_p_rank_array;
	long num_edges = 0;
	adj_vert_t *avert;
	struct timespec time_diff;
	struct timespec diff(struct timespec start, struct timespec end);
	struct timespec time1, time2;

	if(argc != 3)
		return -1;
	num_vertices = string_to_long(argv[1]);
	if(num_vertices < 0)
		return -1;
	graph = (vertex_t *)malloc(num_vertices * sizeof(vertex_t));
	epsilon =(pr_type_t) 0.00001;
	if(!graph)
		return -1;
	initialize_vertices(graph, num_vertices);
	file = fopen(argv[2],"r");
	if(file)
	{
		while (fgets(line, sizeof(line), file))
		{
			token1 = strtok (line,GRAPH_FILE_SEPERATOR);
			token2 = strtok(NULL,GRAPH_FILE_SEPERATOR);
			if(token1 == NULL || token2 == NULL || strtok(NULL,GRAPH_FILE_SEPERATOR) != NULL)
				return -1;
			pnode = string_to_long(token1);
			cnode = string_to_long(token2);
			if(pnode < 0 || cnode < 0)
				return -1;
			if(append_node(graph,pnode,cnode,num_vertices,&num_edges))
				return -1;
		}
	}
	else
		return -1;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Record event on the default stream
	hipEventRecord(start, 0);
	
	//Compact Adjacency list
	vertex_array = (compact_adj_node_t *)calloc(num_vertices, sizeof(compact_adj_node_t));
	edge_array = (long *)calloc(num_edges, sizeof(long));
	p_rank_array = (p_rank_struct_t *)malloc(num_vertices * sizeof(p_rank_struct_t));
	if(hipMalloc(&d_vertex_array,num_vertices * sizeof(compact_adj_node_t)) != hipSuccess)
	{
		printf("Error in cudaMalloc\n");
		return -2;
	}
	if(hipMalloc(&d_edge_array,num_edges * sizeof(long)) != hipSuccess)
	{
		printf("Error in cudaMalloc\n");
		return -2;
	}
	if(hipMalloc(&d_p_rank_array,num_vertices * sizeof(p_rank_struct_t)) != hipSuccess)
	{
		printf("Error in cudaMalloc\n");
		return -2;
	}
	for(i = 0,j = 0;i < num_vertices;i++)
	{
		//Initialize Page Rank values
		p_rank_array[i].next = RAND_HOP_LIKELIHOOD(rand_hop,num_vertices);
		p_rank_array[i].curr = 1 / (pr_type_t)num_vertices;

		vertex_array[i].edge_index = j;
		for(avert = (adj_vert_t *)graph[i].next;avert != NULL;avert = avert->next)
			edge_array[j++] = avert->vertex_num;
		if(vertex_array[i].edge_index - j == 0)
			vertex_array[i].is_leaf = 1;
	}
	if(hipMemcpy(d_vertex_array,vertex_array,num_vertices * sizeof(compact_adj_node_t),hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("Error in cudaMemcpy\n");
		return -2;
	}
	if(hipMemcpy(d_edge_array,edge_array,num_edges * sizeof(long),hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("Error in cudaMemcpy\n");
		return -2;
	}
	if(hipMemcpy(d_p_rank_array,p_rank_array,num_vertices * sizeof(p_rank_struct_t),hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("Error in cudaMemcpy\n");
		return -2;
	}

	printf("Graph parsing successful\n");
	
	if(hipHostMalloc(&d_pr_diff,sizeof(pr_type_t), hipHostMallocDefault) != hipSuccess)
	{
		printf("Error in cudaMalloc\n");
		return -2;
	}

	do
	{
		pr_diff = 0;
		if(hipMemcpy(d_pr_diff,&pr_diff,sizeof(pr_type_t),hipMemcpyHostToDevice) != hipSuccess)
		{
			printf("Error in cudaMemcpy\n");
			return -2;
		}
		calc_bfs_pg_rank_cuda<<<(num_vertices / GPU_BLOCK_SIZE) + 1, GPU_BLOCK_SIZE>>>(d_vertex_array,d_edge_array,d_p_rank_array,num_vertices,num_edges);
		hipDeviceSynchronize();
		update_pr<<<(num_vertices / GPU_BLOCK_SIZE) + 1, GPU_BLOCK_SIZE>>>(d_p_rank_array, d_pr_diff, num_vertices);
		hipDeviceSynchronize();
		if(hipMemcpy(&pr_diff,d_pr_diff,sizeof(pr_type_t),hipMemcpyDeviceToHost) != hipSuccess)
		{
			printf("Error in cudaMemcpy\n");
			return -2;
		}
		
	}while(pr_diff > epsilon);
	if(hipMemcpy(p_rank_array,d_p_rank_array,num_vertices * sizeof(p_rank_struct_t),hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("Error in cudaMemcpy\n");
		return -2;
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_gpu, start, stop);
	printf("\nGPU time: %f (msec)\n", elapsed_gpu);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	//print_converged_pr_vals(p_rank_array, num_vertices);

	free(vertex_array);
	free(edge_array);
	free(p_rank_array);

	hipFree(vertex_array);
	hipFree(edge_array);
	hipHostFree(p_rank_array);
	return 0;
}
